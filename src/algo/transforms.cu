#include "hip/hip_runtime.h"
#include <operations.cuh>

void __matSqrtLongLong(long long *A, long long *B, int m, int n, uint8_t* error) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    if (*error = cuda_fmt_error(hipMalloc((void **) &gpu, sizeof(long long) * N * 2)))
    {
        hipFree(gpu);
        return;
    }

    if (*error = cuda_fmt_error(hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice)))
    {
        hipFree(gpu);
        return;
    }

    const int BLOCK_SIZE = 256;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_sqrt_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, N);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu);
        return;
    }
    hipFree(gpu);
}


void __matExpLongLong(long long *A, long long *B, int m, int n, uint8_t* error) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    if (*error = cuda_fmt_error(hipMalloc((void **) &gpu, sizeof(long long) * N * 2)))
    {
        hipFree(gpu);
        return;
    }

    if (*error = cuda_fmt_error(hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice)))
    {
        hipFree(gpu);
        return;
    }

    const int BLOCK_SIZE = 256;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_exp_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, N);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu);
        return;
    }

    hipFree(gpu);
}



void __zScore(long long* inp, long long* out, long long eps, int n, uint8_t* error)
{
    long long* d_gpu = nullptr;

    if (*error = cuda_fmt_error(hipMalloc(&d_gpu, 2 * n * sizeof(long long))))
    {
        hipFree(d_gpu);
        return;
    }

    if (*error = cuda_fmt_error(hipMemcpy(d_gpu, inp, n * sizeof(long long), hipMemcpyHostToDevice)))
    {
        hipFree(d_gpu);
        return;
    }

    long long mean = __meanReduction_impl(d_gpu, n, error);
    long long std = __stdReduction_impl(d_gpu, n, error);

    if (*error)
    {
        hipFree(d_gpu);
        return;
    }

    const int BLOCK_SIZE = 256;
    zScore_kernel<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_gpu, d_gpu + n, mean, std + eps, n);
    *error = cuda_fmt_error(hipMemcpy(out, d_gpu + n, n * sizeof(long long), hipMemcpyDeviceToHost));
    hipFree(d_gpu);
}


void __maxMinScale(long long* inp, long long* out, int n, uint8_t* error)
{
    long long* d_gpu = nullptr;

    if (*error = cuda_fmt_error(hipMalloc(&d_gpu, 2 * n * sizeof(long long))))
    {
        hipFree(d_gpu);
        return;
    }

    if (*error = cuda_fmt_error(hipMemcpy(d_gpu, inp, n * sizeof(long long), hipMemcpyHostToDevice)))
    {
        hipFree(d_gpu);
        return;
    }

    long long min = __minReduction_impl(d_gpu, n, error);
    long long max = __maxReduction_impl(d_gpu, n, error);

    if (*error)
    {
        hipFree(d_gpu);
        return;
    }

    const int BLOCK_SIZE = 256;
    minMaxScale_kernel<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_gpu, d_gpu + n, min, max, n);
    *error = cuda_fmt_error(hipMemcpy(out, d_gpu + n, n * sizeof(long long), hipMemcpyDeviceToHost));
    hipFree(d_gpu);
}
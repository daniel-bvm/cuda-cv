#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

#include <fixedlonglong32x32.cuh>
#include <kernels.cuh>
#include <operations.cuh>

void __layerNormalizeFixedLongLong(
    long long *X, // input of shape (m, n, c) 
    long long *Y, // output of shape (m, n, c)
    long long *ma, // moving average (c)
    long long *mv, // mong variance  (c)
    long long *gamma, // scale (c)
    long long *beta, // offset (c)
    long long epsilon, // epsilon 
    int h, int w, int c  // m, n, c 
    , bool& error
) 
{
    
}

void __batchNormalizeFixedLongLong(
    long long *X, // input of shape (m, n, c) 
    long long *Y, // output of shape (m, n, c)
    long long *ma, // moving average (c)
    long long *mv, // mong variance  (c)
    long long *gamma, // scale (c)
    long long *beta, // offset (c)
    long long epsilon, // epsilon 
    int h, int w, int c  // m, n, c 
    , bool& error
) 
{
    long long *gpu;
    hipMalloc(&gpu, (h * w * c * 2 + 4 * c) * sizeof(long long));
    hipMemcpy(gpu, X, h * w * c * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(gpu + h * w * c * 2, ma, c * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(gpu + h * w * c * 2 + c, mv, c * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(gpu + h * w * c * 2 + c * 2, gamma, c * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(gpu + h * w * c * 2 + c * 3, beta, c * sizeof(long long), hipMemcpyHostToDevice);

    const dim3 BLOCK_SIZE(32, 32, 1);
    const dim3 GRID_SIZE(
        (w + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x, 
        (h + BLOCK_SIZE.y - 1) / BLOCK_SIZE.y, 
        c
    );

    normalizeFixedLongLong_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(
        gpu, // input of shape (m, n, c)
        gpu + h * w * c,  // output of shape (m, n, c)
        gpu + h * w * c * 2,  // moving average (c)
        gpu + h * w * c * 2 + c,  // moving variance  (c)
        gpu + h * w * c * 2 + c * 2, // gamma (c)
        gpu + h * w * c * 2 + c * 3, // beta (c)
        epsilon, 
        h, w, c
    );

    hipMemcpy(Y, gpu + h * w * c, h * w * c * sizeof(long long), hipMemcpyDeviceToHost);
    hipFree(gpu);
}
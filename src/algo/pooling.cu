#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <cstring>

#include <fixedlonglong32x32.cuh>
#include <operations.cuh>
#include <kernels.cuh>


////////////////////// implementation ///////////////////////// 


void globalAvgPoolingFixedLongLong_impl(
    long long* inp, long long* out,
    int h, int w, int in_channel
    , uint8_t* error
)
{
    const int block_sz = 512;
    const int block_sz2 = block_sz * 2;
    const int grid_sz_x = (h * w + block_sz2 - 1) / block_sz2;
    const int grid_sz_z = in_channel;

    long long* blockSum;
    hipMalloc(&blockSum, sizeof(long long) * grid_sz_x * grid_sz_z);

    sumReductionV2_kernel<<<dim3(grid_sz_x, 1, grid_sz_z), block_sz, sizeof(long long) * block_sz2>>>(
        inp, blockSum, h * w, in_channel
    );

    if (grid_sz_x > 1)
    {
        globalAvgPoolingFixedLongLong_impl(
            blockSum, out, grid_sz_x, 1, in_channel, error
        );
    }
    else
    {
        hipMemcpy(out, blockSum, in_channel * sizeof(long long), hipMemcpyDeviceToDevice);
    }

    hipFree(blockSum);    
}

void __maxPoolingFixedLongLong(
    long long* inp, long long* out, // data io
    int h, int w, int in_channel, // in spatial size, in_channel
    int pool_size, int stride_h, int stride_w, // pooling size, stride 
    int padding // padding mode, one of 'valid': 0 or 'same': 1
    , uint8_t* error
)
{
    // only support for squared pool size, squared input

    if (w != h)
    {
        // not sure if it works
        return;
    }

    if (stride_h <= 0)
    {
        stride_h = pool_size;
    }

    if (stride_w <= 0)
    {
        stride_w = pool_size;
    }

    int pad_top = 0, pad_bottom = 0, pad_left = 0, pad_right = 0;

    if (padding == 1)
    {
        int out_h = (h + stride_h - 1) / stride_h;
        int out_w = (w + stride_w - 1) / stride_w;

        int pad_h = max((out_h - 1) * stride_h + pool_size - h, 0);
        int pad_w = max((out_w - 1) * stride_w + pool_size - w, 0);
        
        pad_top = pad_h / 2;
        pad_bottom = pad_h - pad_top;
        
        pad_left = pad_w / 2;
        pad_right = pad_w - pad_left;
    }

    long long *d_gpu;

    int out_w = (w + pad_left + pad_right - pool_size) / stride_w + 1;
    int out_h = (h + pad_top + pad_bottom - pool_size) / stride_h + 1;

    uint64_t inpFlatSize = (w + pad_left + pad_right) * (h + pad_top + pad_bottom) * in_channel;
    uint64_t outFlatSize = out_h * out_w * in_channel;
    uint64_t flatSize = inpFlatSize + outFlatSize;

    hipMalloc(&d_gpu, flatSize * sizeof(long long));
    
    if (padding == 1)
    {
        long long* padded_inp = new long long[inpFlatSize];
        memset(padded_inp, 0x00, inpFlatSize * sizeof(long long));

        for (int i = 0; i < h; ++i)
        {
            for (int j = 0; j < w; ++j)
            {
                memcpy(padded_inp + ((i + pad_top) * (w + pad_left + pad_right) + j + pad_left) * in_channel, inp + (i * w + j) * in_channel, in_channel << 3);
            }
        }

        hipMemcpy(d_gpu, padded_inp, inpFlatSize * sizeof(long long), hipMemcpyHostToDevice);
        delete[] padded_inp;
    }
    else
    {
        hipMemcpy(d_gpu, inp, inpFlatSize * sizeof(long long), hipMemcpyHostToDevice);
    }
    
    const int thread_x = 32, thread_y=32, thread_z=1; 
    dim3 threads_per_block(thread_x, thread_y, thread_z);

    const dim3 block_per_grid(
        (out_w + threads_per_block.x - 1) / threads_per_block.x,
        (out_h + threads_per_block.y - 1) / threads_per_block.y,
        (in_channel + threads_per_block.z - 1) / threads_per_block.z
    );

    maxPoolingImplFixedLongLong_kernel<<<block_per_grid, threads_per_block>>>(
        d_gpu, d_gpu + inpFlatSize, 
        h + pad_bottom + pad_top, w + pad_left + pad_right, in_channel, 
        pool_size, stride_h, stride_w, 
        out_h, out_w, 
        pad_top, pad_bottom, pad_left, pad_right
    );

    hipMemcpy(out, d_gpu + inpFlatSize, outFlatSize * sizeof(long long), hipMemcpyDeviceToHost);

    hipFree(d_gpu);
}

void __avgPoolingFixedLongLong(
    long long* inp, long long* out,
    int h, int w, int in_channel,
    int pool_size, int stride_h, int stride_w,
    int padding
    , uint8_t* error
)
{
    // only support for squared pool size, squared input

    if (w != h)
    {
        // not sure if it works
        return;
    }

    if (stride_h <= 0)
    {
        stride_h = pool_size;
    }

    if (stride_w <= 0)
    {
        stride_w = pool_size;
    }

    int pad_top = 0, pad_bottom = 0, pad_left = 0, pad_right = 0;

    if (padding == 1)
    {
        int out_h = (h + stride_h - 1) / stride_h;
        int out_w = (w + stride_w - 1) / stride_w;

        int pad_h = max((out_h - 1) * stride_h + pool_size - h, 0);
        int pad_w = max((out_w - 1) * stride_w + pool_size - w, 0);
        
        pad_top = pad_h / 2;
        pad_bottom = pad_h - pad_top;
        
        pad_left = pad_w / 2;
        pad_right = pad_w - pad_left;
    }

    long long *d_gpu;

    int out_w = (w + pad_left + pad_right - pool_size) / stride_w + 1;
    int out_h = (h + pad_top + pad_bottom - pool_size) / stride_h + 1;

    uint64_t inpFlatSize = (w + pad_left + pad_right) * (h + pad_top + pad_bottom) * in_channel;
    uint64_t outFlatSize = out_h * out_w * in_channel;
    uint64_t flatSize = inpFlatSize + outFlatSize;

    hipMalloc(&d_gpu, flatSize * sizeof(long long));
    
    if (padding == 1)
    {
        long long* padded_inp = new long long[inpFlatSize];
        memset(padded_inp, 0x00, inpFlatSize * sizeof(long long));

        for (int i = 0; i < h; ++i)
        {
            for (int j = 0; j < w; ++j)
            {
                memcpy(
                    padded_inp + ((i + pad_top) * (w + pad_left + pad_right) + j + pad_left) * in_channel, 
                    inp + (i * w + j) * in_channel, 
                    in_channel << 3
                );
            }
        }

        hipMemcpy(d_gpu, padded_inp, inpFlatSize * sizeof(long long), hipMemcpyHostToDevice);
        delete[] padded_inp;
    }
    else
    {
        hipMemcpy(d_gpu, inp, inpFlatSize * sizeof(long long), hipMemcpyHostToDevice);
    }

    const int thread_x = 32, thread_y = 32, thread_z = 1;
    dim3 threads_per_block(thread_x, thread_y, thread_z);

    const dim3 block_per_grid(
        (out_w + threads_per_block.x - 1) / threads_per_block.x,
        (out_h + threads_per_block.y - 1) / threads_per_block.y,
        (in_channel + threads_per_block.z - 1) / threads_per_block.z
    );

    avgPoolingImplFixedLongLong_kernel<<<block_per_grid, threads_per_block>>>(
        d_gpu, d_gpu + inpFlatSize, 
        h + pad_top + pad_bottom, w + pad_left + pad_right, in_channel, 
        pool_size, stride_h, stride_w, 
        out_h, out_w,
        pad_top, pad_bottom, pad_left, pad_right
    );

    hipMemcpy(out, d_gpu + inpFlatSize, outFlatSize * sizeof(long long), hipMemcpyDeviceToHost);
    hipFree(d_gpu);
}

void __globalAvgPoolingFixedLongLong(
    long long* inp, long long* out,
    int h, int w, int in_channel,
    uint8_t* error
)
{
    long long* gpu;
    hipMalloc(&gpu, sizeof(long long) * (h * w * in_channel + in_channel));
    hipMemcpy(gpu + in_channel, inp, h * w * in_channel * sizeof(long long), hipMemcpyHostToDevice);
    globalAvgPoolingFixedLongLong_impl(gpu + in_channel, gpu, h, w, in_channel, error);
    hipMemcpy(out, gpu, in_channel * sizeof(long long), hipMemcpyDeviceToHost);

    // assume the number of channel is not too large at the moment
    for (int i = 0; i < in_channel; ++i)
    {
        out[i] = FixedLongLong::div(
            out[i], 
            FixedLongLong::mul(FixedLongLong::fromInt(h), FixedLongLong::fromInt(w))
        );
    }

    hipFree(gpu);
}

// @deprecated
uint8_t estimatePoolingOutputSize(
    int h, int w, int in_channel,
    int pool_size, int padding, 
    int stride_h, int stride_w,
    int* out_h, int* out_w
)
{
    if (!out_h || !out_w)
    {
        return ERROR;
    }

    if (stride_h <= 0)
    {
        stride_h = pool_size;
    }

    if (stride_w <= 0)
    {
        stride_w = pool_size;
    }

    int pad_top = 0, pad_bottom = 0, pad_left = 0, pad_right = 0;

    if (padding == 1)
    {
        int out_h = (h + stride_h - 1) / stride_h;
        int out_w = (w + stride_w - 1) / stride_w;

        int pad_h = max((out_h - 1) * stride_h + pool_size - h, 0);
        int pad_w = max((out_w - 1) * stride_w + pool_size - w, 0);
        
        pad_top = pad_h / 2;
        pad_bottom = pad_h - pad_top;
        
        pad_left = pad_w / 2;
        pad_right = pad_w - pad_left;
    }

    *out_w = (w + pad_left + pad_right - pool_size) / stride_w + 1;
    *out_h = (h + pad_top + pad_bottom - pool_size) / stride_h + 1;
    return OK;
}

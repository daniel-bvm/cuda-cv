#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <cstring>

#include <fixedlonglong32x32.cuh>
#include <kernels.cuh>
#include <operations.cuh>

long long sumReduction_impl(long long* d_gpu, int n, bool& error)
{
    long long res = 0;
    int block_sz = 512;
    int block_sz2 = block_sz * 2;
    int grid_sz = (n + block_sz2 - 1) / block_sz2;

    long long* blockSum;
    hipMalloc(&blockSum, grid_sz * sizeof(long long));
    hipMemset(blockSum, 0, grid_sz * sizeof(long long));

    sumReduction_kernel<<<grid_sz, block_sz, block_sz2 * sizeof(long long)>>>(d_gpu, blockSum, n);

    if (grid_sz > 1)
    {
        res = sumReduction_impl(blockSum, grid_sz, error);
    }
    else
    {
        hipMemcpy(&res, blockSum, sizeof(long long), hipMemcpyDeviceToHost);
    }

    hipFree(blockSum);
    return res;
}



long long maxReduction_impl(long long* d_gpu, int n, bool& error)
{
    long long res = 0;
    int block_sz = 1024;
    int block_sz2 = block_sz * 2;
    int grid_sz = (n + block_sz2 - 1) / block_sz2;

    long long* blockMax;
    hipMalloc(&blockMax, grid_sz * sizeof(long long));
    hipMemset(blockMax, 0, grid_sz * sizeof(long long));

    maxReduction_kernel<<<grid_sz, block_sz, block_sz2 * sizeof(long long)>>>(d_gpu, blockMax, n);

    if (grid_sz > 1)
    {
        res = maxReduction_impl(blockMax, grid_sz, error);
    }
    else
    {
        hipMemcpy(&res, blockMax, sizeof(long long), hipMemcpyDeviceToHost);
    }

    hipFree(blockMax);
    return res;
}



long long minReduction_impl(long long* d_gpu, int n, bool& error)
{
    long long res = 0;
    int block_sz = 512;
    int block_sz2 = block_sz * 2;
    int grid_sz = (n + block_sz2 - 1) / block_sz2;

    long long* blockMin;
    hipMalloc(&blockMin, grid_sz * sizeof(long long));
    hipMemset(blockMin, 0, grid_sz * sizeof(long long));

    sumReduction_kernel<<<grid_sz, block_sz, block_sz2 * sizeof(long long)>>>(d_gpu, blockMin, n);

    if (grid_sz > 1)
    {
        res = minReduction_impl(blockMin, grid_sz, error);
    }
    else
    {
        hipMemcpy(&res, blockMin, sizeof(long long), hipMemcpyDeviceToHost);
    }

    hipFree(blockMin);
    return res;
}

long long __sumReduction(long long* inp, int n, bool& error)
{
    long long* gpu; 
    hipMalloc(&gpu, n * sizeof(long long));
    hipMemcpy(gpu, inp, n * sizeof(long long), hipMemcpyHostToDevice);
    long long res = sumReduction_impl(gpu, n, error);
    hipFree(gpu);
    return res;
}

long long __avgReduction(long long* inp, int n, bool& error)
{
    return FixedLongLong::div(__sumReduction(inp, n, error), (1ll * n) << 32);
}

long long __maxReduction(long long* inp, int n, bool& error)
{
    long long* gpu; 
    hipMalloc(&gpu, n * sizeof(long long));
    hipMemcpy(gpu, inp, n * sizeof(long long), hipMemcpyHostToDevice);
    long long res = maxReduction_impl(gpu, n, error);
    hipFree(gpu);
    return res;
}

long long __minReduction(long long* inp, int n, bool& error)
{
    long long* gpu; 
    hipMalloc(&gpu, n * sizeof(long long));
    hipMemcpy(gpu, inp, n * sizeof(long long), hipMemcpyHostToDevice);
    long long res = minReduction_impl(gpu, n, error);
    hipFree(gpu);
    return res;
}

long long __meanReduction(long long* inp, int n, bool& error)
{
    return FixedLongLong::div(__sumReduction(inp, n, error), (1LL * n) << 32);
}

long long __stdReduction(long long* inp, int n, bool& error)
{
    long long mean = __meanReduction(inp, n, error);
    return 0;
}

void __maxMinScale(long long* inp, long long* out, int n, bool& error)
{
    long long min = __minReduction(inp, n, error);
    long long max = __maxReduction(inp, n, error);

}

void __zScore(long long* inp, long long* out, long long eps, int n, bool& error)
{
    long long mean = __meanReduction(inp, n, error); 
    long long std = __stdReduction(inp, n, error);
}

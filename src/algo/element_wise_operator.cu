#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

#include <fixedlonglong32x32.cuh>
#include <kernels.cuh>
#include <operations.cuh>


////////////////////// kernels ///////////////////////// 


////////////////////// implementations ///////////////////////// 

void __matAddLongLong(long long *A, long long *B, long long *C, int m, int n, uint8_t* error) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    hipMalloc((void **) &gpu, sizeof(long long) * N * 3);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu + N, B, sizeof(long long) * N, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_add_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, gpu + 2 * N, N);

    hipMemcpy(C, gpu + 2 * N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}

void __matSubLongLong(long long *A, long long *B, long long *C, int m, int n, uint8_t* error) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    hipMalloc((void **) &gpu, sizeof(long long) * N * 3);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu + N, B, sizeof(long long) * N, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_sub_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, gpu + 2 * N, N);

    hipMemcpy(C, gpu + 2 * N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
    
}

void __matMulLongLong(long long *A, long long *B, long long *C, int m, int n, uint8_t* error) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    hipMalloc((void **) &gpu, sizeof(long long) * N * 3);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu + N, B, sizeof(long long) * N, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_mul_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, gpu + 2 * N, N);

    hipMemcpy(C, gpu + 2 * N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}

void __matDivLongLong(long long *A, long long *B, long long *C, int m, int n, uint8_t* error) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    hipMalloc((void **) &gpu, sizeof(long long) * N * 3);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu + N, B, sizeof(long long) * N, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_div_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, gpu + 2 * N, N);

    hipMemcpy(C, gpu + 2 * N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}


void __matSqrtLongLong(long long *A, long long *B, int m, int n, uint8_t* error) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    hipMalloc((void **) &gpu, sizeof(long long) * N * 2);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_sqrt_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, N);

    hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}
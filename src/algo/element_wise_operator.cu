#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

#include <fixedlonglong32x32.cuh>
#include <operations.h>
#include <kernels.cuh>


////////////////////// kernels ///////////////////////// 


////////////////////// implementations ///////////////////////// 

void matAddLongLong(long long *A, long long *B, long long *C, int m, int n) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    hipMalloc((void **) &gpu, sizeof(long long) * N * 3);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu + N, B, sizeof(long long) * N, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_add_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, gpu + 2 * N, N);

    hipMemcpy(C, gpu + 2 * N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}

void matSubLongLong(long long *A, long long *B, long long *C, int m, int n) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    hipMalloc((void **) &gpu, sizeof(long long) * N * 3);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu + N, B, sizeof(long long) * N, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_sub_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, gpu + 2 * N, N);

    hipMemcpy(C, gpu + 2 * N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}

void matMulLongLong(long long *A, long long *B, long long *C, int m, int n) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    hipMalloc((void **) &gpu, sizeof(long long) * N * 3);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu + N, B, sizeof(long long) * N, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_mul_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, gpu + 2 * N, N);

    hipMemcpy(C, gpu + 2 * N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}

void matDivLongLong(long long *A, long long *B, long long *C, int m, int n) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    hipMalloc((void **) &gpu, sizeof(long long) * N * 3);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu + N, B, sizeof(long long) * N, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_div_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, gpu + 2 * N, N);

    hipMemcpy(C, gpu + 2 * N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}


void matSqrtLongLong(long long *A, long long *B, int m, int n) {
    // Allocate device memory:
    long long *gpu;
    const int N = m * n;

    hipMalloc((void **) &gpu, sizeof(long long) * N * 2);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    mat_sqrt_fixed_longlong<<<GRID_SIZE, BLOCK_SIZE>>>(gpu, gpu + N, N);

    hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}
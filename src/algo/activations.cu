#include "hip/hip_runtime.h"
#include <fixedlonglong32x32.cuh>
#include <operations.cuh>

// softmax interface
void __softmaxFixedLongLong(long long *A, long long* B, int m, uint8_t* error) 
{
    long long *gpu_a, *gpu_b, *buffer_tmp;

    
    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_a, sizeof(long long)*m)))
    {
        hipFree(gpu_a);
        return;
    }

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_b, sizeof(long long)*m)))
    {
        hipFree(gpu_b);
        return;
    }
    
    buffer_tmp = new long long[m];

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;

    if (*error = cuda_fmt_error(hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice)))
    {
        hipFree(gpu_a);
        delete[] buffer_tmp;
        return;
    }

    const int sqrt_m = sqrt(m);
    const int BUCKETS = (m + sqrt_m - 1) / sqrt_m;
    arrayExp_kernel<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);
    arraySum_kernel<<<BUCKETS, 1>>>(gpu_b, gpu_a, m);
    
    long long sumExp = 0;
    
    if (*error = cuda_fmt_error(hipMemcpy(buffer_tmp, gpu_a, sizeof(long long)*BUCKETS, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu_a);
        delete[] buffer_tmp;
        return;
    }

    for (int i = 0; i < BUCKETS; ++i)
    {
        sumExp += buffer_tmp[i];
    }

    softmaxImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_b, gpu_a, m, sumExp);
    if (*error = cuda_fmt_error(hipMemcpy(B, gpu_a, sizeof(long long)*m, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu_a);
        delete[] buffer_tmp;
        return;
    }
    hipFree(gpu_a), hipFree(gpu_b);
    delete[] buffer_tmp;
}

// sigmoid interface
void __sigmoidFixedLongLong(long long *A, long long* B, int m, uint8_t* error) 
{  
    long long *gpu_a, *gpu_b;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_a, sizeof(long long)*m)))
    {
        hipFree(gpu_a);
        return;
    }

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_b, sizeof(long long)*m)))
    {
        hipFree(gpu_b);
        return;
    }

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
 
    if (*error = cuda_fmt_error(hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice)))
    {
        hipFree(gpu_a);
        return;
    }
    sigmoidImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu_b, sizeof(long long)*m, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu_b);
        return;
    }
    hipFree(gpu_a), hipFree(gpu_b);
}

// tanh interface
void __tanhFixedLongLong(long long *A, long long *B, int m, uint8_t* error) 
{
    long long *gpu_a, *gpu_b;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_a, sizeof(long long)*m)))
    {
        hipFree(gpu_a);
        return;
    }

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_b, sizeof(long long)*m)))
    {
        hipFree(gpu_b);
        return;
    }

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;

    if (*error = cuda_fmt_error(hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice)))
    {
        hipFree(gpu_a);
        return;
    }
    tanhImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu_b, sizeof(long long)*m, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu_b);
        return;
    }
    hipFree(gpu_a), hipFree(gpu_b);
}

// relu interface
void __reluFixedLongLong(long long *A, long long *B, int m, uint8_t* error) 
{
    long long *gpu_a, *gpu_b;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_a, sizeof(long long)*m)))
    {
        hipFree(gpu_a);
        return;
    }

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_b, sizeof(long long)*m)))
    {
        hipFree(gpu_b);
        return;
    }

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;

    if (*error = cuda_fmt_error(hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice)))
    {
        hipFree(gpu_a);
        return;
    }
    reluImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu_b, sizeof(long long)*m, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu_b);
        return;
    }
    hipFree(gpu_a), hipFree(gpu_b);
}

// relu interface
void __relu3DFixedLongLong(long long *A, long long *B, int h, int w, int c, uint8_t* error) 
{
    long long* gpu;
    const int N = h * w * c;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu, sizeof(long long) * N * 2)))
    {
        hipFree(gpu);
        return;
    }

    if (*error = cuda_fmt_error(hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice)))
    {
        hipFree(gpu);
        return;
    }
    
    const dim3 BLOCK_SIZE(256);
    const dim3 BLOCKS((N + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x);
    reluImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + N, N);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu);
        return;
    }
    hipFree(gpu);
}

// relu interface
void __sigmoid3DFixedLongLong(long long *A, long long *B, int h, int w, int c, uint8_t* error) 
{
    long long* gpu;
    const int N = h * w * c;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu, sizeof(long long) * N * 2)))
    {
        hipFree(gpu);
        return;
    }

    if (*error = cuda_fmt_error(hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice)))
    {
        hipFree(gpu);
        return;
    }

    const dim3 BLOCK_SIZE(256);
    const dim3 BLOCKS((N + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x);
    sigmoidImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + N, N);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu);
        return;
    }
    hipFree(gpu);
}


// relu interface
void __tanh3DFixedLongLong(long long *A, long long *B, int h, int w, int c, uint8_t* error) 
{
    long long* gpu;
    const int N = h * w * c;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu, sizeof(long long) * N * 2)))
    {
        hipFree(gpu);
        return;
    }

    if (*error = cuda_fmt_error(hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice)))
    {
        hipFree(gpu);
        return;
    }

    const dim3 BLOCK_SIZE(256);
    const dim3 BLOCKS((N + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x);
    tanhImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + N, N);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu);
        return;
    }
    hipFree(gpu);
}

void __softmax2DFixedLongLong(long long* A, long long* B, int h, int w, int c, uint8_t* error)
{
    memset(B, 0, sizeof(long long) * h * w * c);
}
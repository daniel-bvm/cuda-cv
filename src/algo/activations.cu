#include "hip/hip_runtime.h"
#include <fixedlonglong32x32.cuh>
#include <operations.cuh>

// softmax interface
void __softmaxFixedLongLong(long long *A, long long* B, int m, uint8_t* error) 
{
    if (!m)
    {
        *error = 1;
        return;
    }

    long long *gpu;
    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;

    if (*error = (cuda_fmt_error(hipMalloc((void**)&gpu, sizeof(long long) * m * 2)) 
        || cuda_fmt_error(hipMemcpy(gpu, A, sizeof(long long) * m, hipMemcpyHostToDevice))))
    {
        hipFree(gpu);
        return;
    }
    
    long long mx = __maxReduction_impl(gpu, m, error);
    mat_sub_single_fixed_longlong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + m, mx, m);
    mat_exp_fixed_longlong<<<BLOCKS, BLOCK_SIZE>>>(gpu + m, gpu, m);
    long long sumExp = __sumReduction_impl(gpu, m, error);

    if (!*error && sumExp != 0) {
        softmaxImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + m, m, sumExp);
        *error = cuda_fmt_error(hipMemcpy(B, gpu + m, sizeof(long long) * m, hipMemcpyDeviceToHost));
    }

    hipFree(gpu);
}

// sigmoid interface
void __sigmoidFixedLongLong(long long *A, long long* B, int m, uint8_t* error) 
{  
    long long *gpu_a, *gpu_b;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_a, sizeof(long long)*m)))
    {
        hipFree(gpu_a);
        return;
    }

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_b, sizeof(long long)*m)))
    {
        hipFree(gpu_a), hipFree(gpu_b);
        return;
    }   

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
 
    if (*error = cuda_fmt_error(hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice)))
    {
        hipFree(gpu_a), hipFree(gpu_b);
        return;
    }
    sigmoidImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu_b, sizeof(long long)*m, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu_a), hipFree(gpu_b);
        return;
    }
    hipFree(gpu_a), hipFree(gpu_b);
}

// tanh interface
void __tanhFixedLongLong(long long *A, long long *B, int m, uint8_t* error) 
{
    long long *gpu_a, *gpu_b;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_a, sizeof(long long)*m)))
    {
        hipFree(gpu_a);
        return;
    }

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_b, sizeof(long long)*m)))
    {
        hipFree(gpu_a), hipFree(gpu_b);
        return;
    }

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;

    if (*error = cuda_fmt_error(hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice)))
    {
        hipFree(gpu_a), hipFree(gpu_b);
        return;
    }
    tanhImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu_b, sizeof(long long)*m, hipMemcpyDeviceToHost)));
    hipFree(gpu_a), hipFree(gpu_b);
}

// relu interface
void __reluFixedLongLong(long long *A, long long *B, int m, uint8_t* error) 
{
    long long *gpu_a, *gpu_b;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_a, sizeof(long long)*m)))
    {
        hipFree(gpu_a);
        return;
    }

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu_b, sizeof(long long)*m)))
    {
        hipFree(gpu_a), hipFree(gpu_b);
        return;
    }

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;

    if (*error = cuda_fmt_error(hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice)))
    {
        hipFree(gpu_a), hipFree(gpu_b);
        return;
    }
    reluImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu_b, sizeof(long long)*m, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu_a), hipFree(gpu_b);
        return;
    }
    hipFree(gpu_a), hipFree(gpu_b);
}

// relu interface
void __relu3DFixedLongLong(long long *A, long long *B, int h, int w, int c, uint8_t* error) 
{
    long long* gpu;
    const int N = h * w * c;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu, sizeof(long long) * N * 2)))
    {
        hipFree(gpu);
        return;
    }

    if (*error = cuda_fmt_error(hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice)))
    {
        hipFree(gpu);
        return;
    }
    
    const dim3 BLOCK_SIZE(256);
    const dim3 BLOCKS((N + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x);
    reluImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + N, N);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu);
        return;
    }
    hipFree(gpu);
}

// relu interface
void __sigmoid3DFixedLongLong(long long *A, long long *B, int h, int w, int c, uint8_t* error) 
{
    long long* gpu;
    const int N = h * w * c;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu, sizeof(long long) * N * 2)))
    {
        hipFree(gpu);
        return;
    }

    if (*error = cuda_fmt_error(hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice)))
    {
        hipFree(gpu);
        return;
    }

    const dim3 BLOCK_SIZE(256);
    const dim3 BLOCKS((N + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x);
    sigmoidImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + N, N);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost)))
    {
        hipFree(gpu);
        return;
    }
    hipFree(gpu);
}


// relu interface
void __tanh3DFixedLongLong(long long *A, long long *B, int h, int w, int c, uint8_t* error) 
{
    long long* gpu;
    const int N = h * w * c;

    if (*error = cuda_fmt_error(hipMalloc((void**)&gpu, sizeof(long long) * N * 2)))
    {
        hipFree(gpu);
        return;
    }

    if (*error = cuda_fmt_error(hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice)))
    {
        hipFree(gpu);
        return;
    }

    const dim3 BLOCK_SIZE(256);
    const dim3 BLOCKS((N + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x);
    tanhImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + N, N);

    if (*error = cuda_fmt_error(hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost)));
    hipFree(gpu);
}

void __softmax2DFixedLongLong(long long* A, long long* B, int h, int w, int c, uint8_t* error)
{
    memset(B, 0, sizeof(long long) * h * w * c);
}
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

#include <fixedlonglong32x32.cuh>
#include <kernels.cuh>
#include <operations.cuh>

// softmax interface
void __softmaxFixedLongLong(long long *A, long long* B, int m, uint8_t* error) 
{
    long long *gpu_a, *gpu_b, *buffer_tmp;

    hipMalloc((void**)&gpu_a, sizeof(long long)*m);
    hipMalloc((void**)&gpu_b, sizeof(long long)*m);
    buffer_tmp = new long long[m];

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice);

    const int sqrt_m = sqrt(m);
    const int BUCKETS = (m + sqrt_m - 1) / sqrt_m;
    arrayExp_kernel<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);
    arraySum_kernel<<<BUCKETS, 1>>>(gpu_b, gpu_a, m);
    
    long long sumExp = 0;
    hipMemcpy(buffer_tmp, gpu_a, sizeof(long long)*BUCKETS, hipMemcpyDeviceToHost);
    for (int i = 0; i < BUCKETS; ++i)
    {
        sumExp += buffer_tmp[i];
    }

    softmaxImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_b, gpu_a, m, sumExp);
    hipMemcpy(B, gpu_a, sizeof(long long)*m, hipMemcpyDeviceToHost);
    hipFree(gpu_a), hipFree(gpu_b);
    delete[] buffer_tmp;
}

// sigmoid interface
void __sigmoidFixedLongLong(long long *A, long long* B, int m, uint8_t* error) 
{  
    long long *gpu_a, *gpu_b;
    
    hipMalloc((void**)&gpu_a, sizeof(long long)*m);
    hipMalloc((void**)&gpu_b, sizeof(long long)*m);

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice);
    sigmoidImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);

    hipMemcpy(B, gpu_b, sizeof(long long)*m, hipMemcpyDeviceToHost);
    hipFree(gpu_a), hipFree(gpu_b);
}

// tanh interface
void __tanhFixedLongLong(long long *A, long long *B, int m, uint8_t* error) 
{
    long long *gpu_a, *gpu_b;
    
    hipMalloc((void**)&gpu_a, sizeof(long long)*m);
    hipMalloc((void**)&gpu_b, sizeof(long long)*m);

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice);
    tanhImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);

    hipMemcpy(B, gpu_b, sizeof(long long)*m, hipMemcpyDeviceToHost);
    hipFree(gpu_a), hipFree(gpu_b);
}

// relu interface
void __reluFixedLongLong(long long *A, long long *B, int m, uint8_t* error) 
{
    long long *gpu_a, *gpu_b;
    
    hipMalloc((void**)&gpu_a, sizeof(long long)*m);
    hipMalloc((void**)&gpu_b, sizeof(long long)*m);

    const int BLOCK_SIZE = 256;
    const int BLOCKS = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMemcpy(gpu_a, A, sizeof(long long)*m, hipMemcpyHostToDevice);
    reluImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu_a, gpu_b, m);

    hipMemcpy(B, gpu_b, sizeof(long long)*m, hipMemcpyDeviceToHost);
    hipFree(gpu_a), hipFree(gpu_b);
}

// relu interface
void __relu3DFixedLongLong(long long *A, long long *B, int h, int w, int c, uint8_t* error) 
{
    long long* gpu;
    const int N = h * w * c;

    hipMalloc((void**)&gpu, sizeof(long long) * N * 2);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);

    const dim3 BLOCK_SIZE(256);
    const dim3 BLOCKS((N + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x);
    reluImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + N, N);

    hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}

// relu interface
void __sigmoid3DFixedLongLong(long long *A, long long *B, int h, int w, int c, uint8_t* error) 
{
    long long* gpu;
    const int N = h * w * c;

    hipMalloc((void**)&gpu, sizeof(long long) * N * 2);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);

    const dim3 BLOCK_SIZE(256);
    const dim3 BLOCKS((N + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x);
    sigmoidImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + N, N);

    hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}


// relu interface
void __tanh3DFixedLongLong(long long *A, long long *B, int h, int w, int c, uint8_t* error) 
{
    long long* gpu;
    const int N = h * w * c;

    hipMalloc((void**)&gpu, sizeof(long long) * N * 2);
    hipMemcpy(gpu, A, sizeof(long long) * N, hipMemcpyHostToDevice);

    const dim3 BLOCK_SIZE(256);
    const dim3 BLOCKS((N + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x);
    tanhImplFixedLongLong<<<BLOCKS, BLOCK_SIZE>>>(gpu, gpu + N, N);

    hipMemcpy(B, gpu + N, sizeof(long long) * N, hipMemcpyDeviceToHost);
    hipFree(gpu);
}

void __softmax2DFixedLongLong(long long* A, long long* B, int h, int w, int c, uint8_t* error)
{
    memset(B, 0, sizeof(long long) * h * w * c);
}
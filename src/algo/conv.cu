#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <fixedlonglong32x32.cuh>
#include <operations.h>
#include <kernels.cuh>

////////////////////// implementation ///////////////////////// 

void conv2dFixedLongLong(
    long long* inp, long long* kernel, long long* bias, long long* out, // data io
    int kernel_size, int in_channel, int out_channel, // kernel properties
    int h, int w, // spatial size of inp,
    int padding, int stride_h, int stride_w // padding: same(0) or valid(1)
)
{

    // @TODO: reduce redundant iterations and support for k_w != k_h case, in_w != in_h also

    // references:
    // [1] https://www.pico.net/kb/what-is-the-difference-between-same-and-valid-padding-in-tf-nn-max-pool-of-tensorflow/

    // parameters
    // inp: [h, w, in_channel]
    // kernel: [kernel_size, kernel_size, in_channel, out_channel]
    // out: [h_out, w_out, out_channel]
    // kernel_size: size of kernel
    // in_channel: number of input channels
    // out_channel: number of output channels
    // w: width of input
    // h: height of input
    // padding: padding mode, one of 'valid': 0 or 'same': 1
    // strides: strides of kernel along width and height

    if (w != h)
    {
        // not sure if it works
        return;
    }

    int pad_top = 0, pad_bottom = 0, pad_left = 0, pad_right = 0;

    if (padding == 1)
    {
        int out_h = (h + stride_h - 1) / stride_h;
        int out_w = (w + stride_w - 1) / stride_w;

        int pad_h = max((out_h - 1) * stride_h + kernel_size - h, 0);
        int pad_w = max((out_w - 1) * stride_w + kernel_size - w, 0);

        pad_top = pad_h / 2;
        pad_bottom = pad_h - pad_top;

        pad_left = pad_w / 2;
        pad_right = pad_w - pad_left;
    }

    long long *d_gpu;

    const int out_w = (w + pad_left + pad_right - kernel_size) / stride_w + 1;
    const int out_h = (h + pad_top + pad_bottom - kernel_size) / stride_h + 1;

    const uint64_t inpFlatSize = (w + pad_left + pad_right) * (h + pad_top + pad_bottom) * in_channel;
    const uint64_t kernelFlatSize = out_channel * kernel_size * kernel_size * in_channel;
    const uint64_t outFlatSize = out_h * out_w * out_channel;
    const uint64_t cudaMemSize = inpFlatSize + outFlatSize + kernelFlatSize + out_channel;

    hipMalloc(&d_gpu, cudaMemSize * sizeof(long long));

    if (padding == 1)
    {
        long long* padded_inp = new long long[inpFlatSize];
        memset(padded_inp, 0x00, inpFlatSize * sizeof(long long));

        for (int i = 0; i < h; ++i)
        {
            for (int j = 0; j < w; ++j)
            {
                memcpy(padded_inp + ((i + pad_top) * (w + pad_left + pad_right) + j + pad_left) * in_channel, inp + (i * w + j) * in_channel, in_channel << 3);
            }
        }

        hipMemcpy(d_gpu, padded_inp, inpFlatSize * sizeof(long long), hipMemcpyHostToDevice);

        delete[] padded_inp;
    }
    else
    {
        hipMemcpy(d_gpu, inp, inpFlatSize * sizeof(long long), hipMemcpyHostToDevice);
    }
    
    hipMemcpy(d_gpu + inpFlatSize, kernel, kernelFlatSize * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(d_gpu + inpFlatSize + kernelFlatSize, bias, out_channel * sizeof(long long), hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 32;

    const dim3 THREAD_PER_BLOCK(BLOCK_SIZE, BLOCK_SIZE, 1);
    const dim3 BLOCK_PER_GRID((out_w + BLOCK_SIZE - 1) / BLOCK_SIZE, (out_h + BLOCK_SIZE - 1) / BLOCK_SIZE, out_channel);

    conv2dImplFixedLongLong_kernel<<<BLOCK_PER_GRID, THREAD_PER_BLOCK>>>(
        d_gpu, // inp
        d_gpu + inpFlatSize, // kernel 
        d_gpu + inpFlatSize + kernelFlatSize, // bias 
        d_gpu + inpFlatSize + kernelFlatSize + out_channel, // out 
        kernel_size, in_channel, out_channel, 
        w + pad_left + pad_right, h + pad_top + pad_bottom, 
        out_w, out_h, padding, stride_h, stride_w
    );

    hipMemcpy(
        out, d_gpu + inpFlatSize + kernelFlatSize + out_channel, 
        out_h * out_w * out_channel * sizeof(long long), 
        hipMemcpyDeviceToHost
    );

    hipFree(d_gpu);
}     

void estimateConvOutputSize(
    int kernel_size, int in_channel, int out_channel, // kernel properties
    int h, int w, // spatial size of inp,
    int padding, int stride_h, int stride_w, // padding: same(0) or valid(1)
    int& out_h, int& out_w // spatial size of out
)
{
    int pad_top = 0, pad_bottom = 0, pad_left = 0, pad_right = 0;

    if (padding == 1)
    {
        int _out_h = (h + stride_h - 1) / stride_h;
        int _out_w = (w + stride_w - 1) / stride_w;

        int pad_h = max((_out_h - 1) * stride_h + kernel_size - h, 0);
        int pad_w = max((_out_w - 1) * stride_w + kernel_size - w, 0);

        pad_top = pad_h / 2;
        pad_bottom = pad_h - pad_top;

        pad_left = pad_w / 2;
        pad_right = pad_w - pad_left;
    }

    out_w = (w + pad_left + pad_right - kernel_size) / stride_w + 1;
    out_h = (h + pad_top + pad_bottom - kernel_size) / stride_h + 1;
}
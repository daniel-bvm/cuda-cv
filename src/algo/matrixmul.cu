#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include <fixedlonglong32x32.cuh>
#include <operations.h>
#include <kernels.cuh>


void maxmulFloat(float *A, float *B, float *C, int m, int n, int k) {

    // Allocate device memory:
    float *gpu_A;
    float *gpu_B;
    float *gpu_C;

    hipMalloc((void **) &gpu_A, sizeof(float) * m * n);
    hipMemcpy(gpu_A, A, sizeof(float) * m * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &gpu_B, sizeof(float) * n * k);
    hipMemcpy(gpu_B, B, sizeof(float) * n * k, hipMemcpyHostToDevice);

    hipMalloc((void **) &gpu_C, sizeof(float) * m * k);

    int BLOCK_SIZE = 16;
    // Blocks & grids:
    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(grid_cols, grid_rows);

    // Call the kernel:
    vecmulFloat<<<grid, blocks>>>(gpu_A, gpu_B, gpu_C, m, n, k);

    // Get the result Matrix:
    hipMemcpy(C, gpu_C, sizeof(float) * m * k, hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(gpu_A);
    hipFree(gpu_B);
    hipFree(gpu_C);
}

void maxmulFixedLongLong(long long *A, long long *B, long long *C, int m, int n, int k) {
    // cout << "Start maxmulFixedLongLong\n";

    // Allocate device memory:
    long long *gpu;
    hipMallocManaged(&gpu, sizeof(long long) * (m * n + n * k + m * k));
    hipMemcpy(gpu, A, sizeof(long long) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(gpu + n * m, B, sizeof(long long) * n * k, hipMemcpyHostToDevice);

    int BLOCK_SIZE = 32;
    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(grid_cols, grid_rows);

    // Call the kernel:
    vecmulFixedLongLong<<<grid, blocks>>>(
        gpu, 
        gpu + n * m, 
        gpu + m * n + n * k, 
        m, n, k
    );

    hipMemcpy(C, gpu + m * n + n * k, sizeof(long long) * m * k, hipMemcpyDeviceToHost);
    hipFree(gpu);
}

void maxmulLong(long *A, long *B, long *C, long m, long n, long k) {

    // Allocate device memory:
    long *gpu_A;
    long *gpu_B;
    long *gpu_C;

    hipMalloc((void **) &gpu_A, sizeof(long) * m * n);
    hipMemcpy(gpu_A, A, sizeof(long) * m * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &gpu_B, sizeof(long) * n * k);
    hipMemcpy(gpu_B, B, sizeof(long) * n * k, hipMemcpyHostToDevice);

    hipMalloc((void **) &gpu_C, sizeof(long) * m * k);

    int BLOCK_SIZE = 16;
    // Blocks & grids:
    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(grid_cols, grid_rows);

    // Call the kernel:
    vecmulLong<<<grid, blocks>>>(gpu_A, gpu_B, gpu_C, m, n, k);

    // Get the result Matrix:
    hipMemcpy(C, gpu_C, sizeof(long) * m * k, hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(gpu_A);
    hipFree(gpu_B);
    hipFree(gpu_C);
}

void maxmulInt(int *A, int *B, int *C, int m, int n, int k) {

    // Allocate device memory:
    int *gpu_A;
    int *gpu_B;
    int *gpu_C;

    hipMalloc((void **) &gpu_A, sizeof(int) * m * n);
    hipMemcpy(gpu_A, A, sizeof(int) * m * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &gpu_B, sizeof(int) * n * k);
    hipMemcpy(gpu_B, B, sizeof(int) * n * k, hipMemcpyHostToDevice);

    hipMalloc((void **) &gpu_C, sizeof(int) * m * k);

    int BLOCK_SIZE = 16;
    // Blocks & grids:
    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(grid_cols, grid_rows);

    // Call the kernel:
    vecmulInt<<<grid, blocks>>>(gpu_A, gpu_B, gpu_C, m, n, k);

    // Get the result Matrix:
    hipMemcpy(C, gpu_C, sizeof(int) * m * k, hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(gpu_A);
    hipFree(gpu_B);
    hipFree(gpu_C);
}

void maxmulDouble(double *A, double *B, double *C, int m, int n, int k) {

    // Allocate device memory:
    double *gpu_A;
    double *gpu_B;
    double *gpu_C;

    // cout << "------------------\n";
    // cout << "A = {";
    //for (int i = 0; i < m * n; ++i) cout << A[i] << (i != m * n - 1 ? ", " : "}\n");
    hipMalloc((void **) &gpu_A, sizeof(double) * m * n);
    hipMemcpy(gpu_A, A, sizeof(double) * m * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &gpu_B, sizeof(double) * n * k);
    hipMemcpy(gpu_B, B, sizeof(double) * n * k, hipMemcpyHostToDevice);

    hipMalloc((void **) &gpu_C, sizeof(double) * m * k);

    int BLOCK_SIZE = 32;
    // Blocks & grids:
    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(grid_cols, grid_rows);

    // Call the kernel:
    vecmulDouble<<<grid, blocks>>>(gpu_A, gpu_B, gpu_C, m, n, k);

    // Get the result Matrix:
    hipMemcpy(C, gpu_C, sizeof(double) * m * k, hipMemcpyDeviceToHost);
    // cout << "C = {";
    // for (int i = 0; i < m * k; ++i) cout << C[i] << (i != m * k - 1 ? ", " : "}\n");
    //Free device matrices
    hipFree(gpu_A);
    hipFree(gpu_B);
    hipFree(gpu_C);
}
#include "hip/hip_runtime.h"
#include <operations.cuh>
#include <computelib.h>
#include <tensor.h>
#include <memory.h>
#include <iostream>
#include <fstream> 
#include <bitset>
#include <cstring>
#include <vector>
#include <algorithm>
#include <numeric>


void logd(const char* msg)
{
    printf("%s - %s - %s: %s\n", __FILE__, __FUNCTION__, __LINE__, msg);
}

uint8_t* conv2d_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    if (pack.tensors.size() != 3 || pack.tensors[0].shape().size() != 3)
    {
        *_error = true;
        return nullptr;
    }

    const std::vector<int64_t>& params = pack.params;

    const std::vector<uint64_t>& inp = pack.tensors[0].shape(),
                                 kernel = pack.tensors[1].shape(),
                                 bias = pack.tensors[2].shape();

    // inp: [h, w, in_c]
    // kernel: [kh, kw, in_c, out_c]
    // bias: [out_c]
    if (kernel.size() != 4 || bias.size() != 1 || kernel[3] != bias[0] || kernel[2] != inp[2])
    {
        *_error = true;
        return nullptr;
    }

    uint32_t h_in = inp[0], w_in = inp[1], c_in = inp[2], c_out = kernel[3], h_out, w_out;
    uint32_t kh = kernel[0], kw = kernel[1], padding = params[2], stride_h = params[0], stride_w = params[1];
    estimateConvOutputSize(kh, c_in, c_out, h_in, w_in, padding, stride_h, stride_w, (int*) &h_out, (int*) &w_out);

    std::vector<uint64_t> out_shape = {h_out, w_out, c_out};
    int64_t* out = new int64_t[h_out * w_out * c_out];

    __conv2dFixedLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)pack.tensors[1].data(), 
        (long long*)pack.tensors[2].data(), 
        (long long*)out, 
        kw, c_in, c_out, 
        h_in, h_out, padding, 
        stride_h, stride_w, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(out_shape, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* maxpooling2d_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    if (pack.tensors.size() != 1 || pack.tensors[0].shape().size() != 3)
    {
        *_error = true;
        return nullptr;
    }

    const std::vector<int64_t>& params = pack.params;
    const std::vector<uint64_t>& inp = pack.tensors[0].shape();

    if (params.size() != 5)
    {
        *_error = true;
        return nullptr;
    }

    uint32_t h_in = inp[0], w_in = inp[1], c_in = inp[2], h_out, w_out;
    uint32_t kh = params[0], kw = params[1], stride_h = params[2], stride_w = params[3], padding = params[4];

    estimatePoolingOutputSize(
        h_in, w_in, c_in, kh, padding, stride_h, stride_w, (int*)&h_out, (int*)&w_out
    );

    std::vector<uint64_t> out_shape = {h_out, w_out, c_in};
    int64_t* out = new int64_t[h_out * w_out * c_in];

    __maxPoolingFixedLongLong(
        (long long*) pack.tensors[0].data(),
        (long long*) out, 
        h_in, w_in, c_in, kh, 
        stride_h, stride_w, padding, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(out_shape, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* avgpooling2d_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    if (pack.tensors.size() != 1 || pack.tensors[0].shape().size() != 3)
    {
        *_error = true;
        return nullptr;
    }

    const std::vector<int64_t>& params = pack.params;
    const std::vector<uint64_t>& inp = pack.tensors[0].shape();

    if (params.size() != 5)
    {
        *_error = true;
        return nullptr;
    }

    uint32_t h_in = inp[0], w_in = inp[1], c_in = inp[2], h_out, w_out;
    uint32_t kh = params[0], kw = params[1], stride_h = params[2], stride_w = params[3], padding = params[4];

    estimatePoolingOutputSize(
        h_in, w_in, c_in, kh, padding, stride_h, stride_w, (int*)&h_out, (int*)&w_out
    );

    std::vector<uint64_t> out_shape = {h_out, w_out, c_in};
    int64_t* out = new int64_t[h_out * w_out * c_in];

    __avgPoolingFixedLongLong(
        (long long*)pack.tensors[0].data(),
        (long long*)out, h_in, w_in, c_in, kh, 
        stride_h, stride_w, padding, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(out_shape, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* matmul_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    if (pack.tensors.size() != 2)
    {
        *_error = true;
        return nullptr;
    }

    uint32_t h1 = 1, 
        w1 = *pack.tensors[0].shape().rbegin(), 
        h2 = 1, 
        w2 = *pack.tensors[1].shape().rbegin();

    const auto& shape1 = pack.tensors[0].shape();
    const auto& shape2 = pack.tensors[1].shape();

    for (int i = 0; i < shape1.size() - 1; ++i) 
    {
        h1 *= shape1[i];
    }

    for (int i = 0; i < shape2.size() - 1; ++i) 
    {
        h2 *= shape2[i];
    }

    if (w1 != h2)
    {
        *_error = true;
        return nullptr;
    }

    std::vector<uint64_t> out_shape = {h1, w2};
    int64_t* out = new int64_t[h1 * w2];  

    __maxmulFixedLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)pack.tensors[1].data(), 
        (long long*)out, 
        h1, w1, w2, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(out_shape, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* elementwise_add_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    if (pack.tensors.size() != 2)
    {
        *_error = true;
        return nullptr;
    }

    int prod1 = 1, prod2 = 1;
    const std::vector<uint64_t>& s1 = pack.tensors[0].shape(),
                                 s2 = pack.tensors[1].shape();

    for (const int& x: s1)
    {
        prod1 *= x;
    }

    for (const int& x: s2)
    {
        prod2 *= x;
    }

    if (prod1 != prod2)
    {
        *_error = true;
        return nullptr;
    }

    int64_t* out = new int64_t[prod1];
    __matAddLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)pack.tensors[1].data(), 
        (long long*)out, 
        1,
        prod1, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(s1, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* elementwise_mul_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    if (pack.tensors.size() != 2)
    {
        *_error = true;
        return nullptr;
    }

    int prod1 = 1, prod2 = 1;
    const std::vector<uint64_t>& s1 = pack.tensors[0].shape(),
                                 s2 = pack.tensors[1].shape();

    for (const int& x: s1)
    {
        prod1 *= x;
    }

    for (const int& x: s2)
    {
        prod2 *= x;
    }

    if (prod1 != prod2)
    {
        *_error = true;
        return nullptr;
    }

    int64_t* out = new int64_t[prod1];
    __matMulLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)pack.tensors[1].data(), 
        (long long*)out, 
        1,
        prod1, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(s1, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* elementwise_sub_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    if (pack.tensors.size() != 2)
    {
        *_error = true;
        return nullptr;
    }

    int prod1 = 1, prod2 = 1;
    const std::vector<uint64_t>& s1 = pack.tensors[0].shape(),
                                 s2 = pack.tensors[1].shape();

    for (const int& x: s1)
    {
        prod1 *= x;
    }

    for (const int& x: s2)
    {
        prod2 *= x;
    }

    if (prod1 != prod2)
    {
        *_error = true;
        return nullptr;
    }

    int64_t* out = new int64_t[prod1];
    __matSubLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)pack.tensors[1].data(), 
        (long long*)out, 
        1,
        prod1, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(s1, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* elementwise_div_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    if (pack.tensors.size() != 2)
    {
        *_error = true;
        return nullptr;
    }

    int prod1 = 1, prod2 = 1;
    const std::vector<uint64_t>& s1 = pack.tensors[0].shape(),
                                 s2 = pack.tensors[1].shape();

    for (const int& x: s1)
    {
        prod1 *= x;
    }

    for (const int& x: s2)
    {
        prod2 *= x;
    }

    if (prod1 != prod2)
    {
        *_error = true;
        return nullptr;
    }

    int64_t* out = new int64_t[prod1];
    __matDivLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)pack.tensors[1].data(), 
        (long long*)out, 
        1,
        prod1, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(s1, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* transform_exp_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{

}

uint8_t* transform_sqrt_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{

}

uint8_t* batch_norm_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    // inp, ma, mv, gama, beta
    if (pack.tensors.size() != 5)
    {
        *_error = true;
        return nullptr;
    }

    const std::vector<uint64_t>& inp = pack.tensors[0].shape(),
                                 ma = pack.tensors[1].shape(),
                                 mv = pack.tensors[2].shape(),
                                 gama = pack.tensors[3].shape(),
                                 beta = pack.tensors[4].shape();

    const std::vector<int64_t>& params = pack.params;

    const int h_in = inp[0], w_in = inp[1], c_in = inp[2];
    
    if (ma.size() != 1 || mv.size() != 1 || gama.size() != 1 || beta.size() != 1)
    {
        *_error = true;
        return nullptr;
    }

    if (params.size() == 0 || ma[0] != c_in || mv[0] != c_in || gama[0] != c_in || beta[0] != c_in)
    {
        *_error = true;
        return nullptr;
    }

    int64_t* out = new int64_t[h_in * w_in * c_in];
    __batchNormalizeFixedLongLong(
        (long long*)pack.tensors[0].data(), // inp
        (long long*)out, 
        (long long*)pack.tensors[1].data(), // ma
        (long long*)pack.tensors[2].data(), // mv
        (long long*)pack.tensors[3].data(), // gama
        (long long*)pack.tensors[4].data(), // beta 
        params[0], // epsilon
        h_in, w_in, c_in, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(inp, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* layer_norm_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* zscore_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* min_max_scale_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* concatenate_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    int n_tensors = pack.tensors.size();
    
    if (!n_tensors)
    {
        *_error = true;
        return nullptr;
    }

    const std::vector<int64_t>& params = pack.params;

    if (params.size() == 0)
    {
        *_error = true;
        return nullptr;
    }

    int64_t** inp_tensors = new int64_t*[n_tensors];
    int64_t** shapes = new int64_t*[n_tensors];
    int common_dims = pack.params[0];


    for (int i = 0; i < n_tensors; ++i)
    {
        inp_tensors[i] = pack.tensors[i].data();
        shapes[i] = (int64_t*)(&pack.tensors[i].shape()[0]);
    }

    std::vector<uint64_t> out_shape(common_dims, 0);

    if (estimateConcatenate_dummy((long long**)shapes, params[0], common_dims, n_tensors, (long long*)&out_shape[0]))
    {
        *_error = true;
        delete[] inp_tensors;
        delete[] shapes;
        return nullptr;
    }
    
    int32_t prod = 1;

    for (const int& x: out_shape)
    {
        prod *= x;
    }

    int64_t* out = new int64_t[prod];
    __concatenate_dummy(
        (long long**)inp_tensors, 
        (long long*)out, 
        (long long**)shapes, 
        params[0], 
        common_dims, 
        n_tensors,
        _error
    );

    if (*_error)
    {
        delete[] out;
        delete[] inp_tensors;
        delete[] shapes;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(out_shape, out), 
        length_out
    );

    delete[] out;
    delete[] inp_tensors;
    delete[] shapes;

    return out_bytes;
}

uint8_t* relu_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    int n_tensor = pack.tensors.size();
    
    if (n_tensor != 1)
    {
        *_error = true;
        return nullptr;
    }

    const std::vector<uint64_t>& inp = pack.tensors[0].shape();
    uint64_t prod = 1;
    
    for (const int& x: inp)
    {
        prod *= x;
    }

    int64_t* out = new int64_t[prod];
    __reluFixedLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)out, 
        prod, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(inp, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* tanh_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    int n_tensor = pack.tensors.size();
    
    if (n_tensor != 1)
    {
        *_error = true;
        return nullptr;
    }

    const std::vector<uint64_t>& inp = pack.tensors[0].shape();
    uint64_t prod = 1;
    
    for (const int& x: inp)
    {
        prod *= x;
    }

    int64_t* out = new int64_t[prod];
    __tanhFixedLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)out, 
        prod, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(inp, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* sigmoid_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    int n_tensor = pack.tensors.size();
    
    if (n_tensor != 1)
    {
        *_error = true;
        return nullptr;
    }

    const std::vector<uint64_t>& inp = pack.tensors[0].shape();
    uint64_t prod = 1;
    
    for (const int& x: inp)
    {
        prod *= x;
    }

    int64_t* out = new int64_t[prod];
    __sigmoidFixedLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)out, 
        prod, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(inp, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* softmax_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    int n_tensor = pack.tensors.size();
    
    if (n_tensor != 1)
    {
        *_error = true;
        return nullptr;
    }

    const std::vector<uint64_t>& inp = pack.tensors[0].shape();
    uint64_t prod = 1;
    
    for (const int& x: inp)
    {
        prod *= x;
    }

    int64_t* out = new int64_t[prod];
    __softmaxFixedLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)out, 
        prod, 
        _error
    );

    if (*_error)
    {
        delete[] out;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(inp, out), 
        length_out
    );

    delete[] out;
    return out_bytes;
}

uint8_t* logsoftmax_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* softmax2d_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* reduction_max_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* reduction_min_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* reduction_mean_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* reduction_sum_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* reduction_argmax_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* reduction_argmin_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    *_error = true;
    return nullptr;
}

uint8_t* dropout_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    if (pack.tensors.size() != 1)
    {
        *_error = true;
        return nullptr;
    }

    return abi_encode_tensor(pack.tensors[0], length_out);
}

uint8_t* globalavgpooling_call(const operation_pack& pack, int32_t* length_out, uint8_t* _error)
{
    if (pack.tensors.size() != 1 || pack.tensors[0].shape().size() < 3)
    {
        *_error = true;
        return nullptr;
    }

    const std::vector<uint64_t>& inp = pack.tensors[0].shape();
    std::vector<uint64_t> out_shape;
    
    for (int i = 0; i < inp.size() - 3; ++i)
    {
        out_shape.push_back(inp[i]);
    }

    out_shape.push_back(inp.back());
    
    int64_t prod = std::accumulate(out_shape.begin(), out_shape.end(), 1, std::multiplies<int64_t>()); 
    int64_t* buffer = new int64_t[prod];

    __globalAvgPoolingFixedLongLong(
        (long long*)pack.tensors[0].data(), 
        (long long*)buffer, 
        prod / out_shape.back(), 1,
        out_shape.back(), 
        _error
    );

    if (*_error)
    {
        delete[] buffer;
        return nullptr;
    }

    uint8_t* out_bytes = abi_encode_tensor(
        TensorWrapper(out_shape, buffer), 
        length_out
    );

    delete[] buffer;
    return out_bytes;
}


int64_t read_opcode(const int64_t* data, uint8_t *__error)
{
    return data[3];
}

std::vector<int64_t> read_params(const int64_t* data, uint8_t *__error)
{
    const int64_t* data3 = data + 3;
    int64_t params_offset = data3[4] >> 3;
    int32_t n_params = data3[params_offset];
    std::vector<int64_t> params(n_params, 0);

    for (int i = 0; i < n_params; ++i)
    {
        params[i] = data3[params_offset + (i + 1) * 4];
    }

    return params;
}

std::vector<std::vector<uint64_t >> read_shapes(const int64_t* data, uint8_t *__error)
{
    const int64_t* data3 = data + 3;
    int64_t shapes_offset = data3[8] >> 3;
    int32_t n_tensor = data3[shapes_offset]; shapes_offset += 4;
    std::vector<std::vector<uint64_t>> shapes(n_tensor, std::vector<uint64_t>());

    for (int i = 0; i < n_tensor; ++i)
    {
        int offset = shapes_offset + (data3[shapes_offset + (i << 2)] >> 3);
        int ndims = data3[offset]; offset += 4; // count
        shapes[i].resize(ndims);

        for (int j = 0; j < ndims; ++j, offset += 4)
        {
            if (data3[offset] <= 0)
            {
                *__error = true;
                return {};
            }

            shapes[i][j] = data3[offset];
        }
    }

    return shapes;
}

std::vector<TensorWrapper> read_tensors(const int64_t* data, uint8_t *__error)
{
    const int64_t* data3 = data + 3;
    int64_t tensors_offset = data3[12] >> 3;
    std::vector<std::vector<uint64_t>> shapes = read_shapes(data, __error);

    if (*__error || data3[tensors_offset] != shapes.size())
    {
        *__error = true;
        std::cerr << "Error in reading shapes" << *__error << " " <<  data3[tensors_offset] << " " << shapes.size() << std::endl;
        return {};
    }

    tensors_offset += 4;
    std::vector<TensorWrapper> tensors;

    for (int i = 0; i < shapes.size(); ++i)
    {
        int offset = tensors_offset + (data3[tensors_offset + (i << 2)] >> 3);
        int cnt = data3[offset]; offset += 4;

        int prod = 1;
        for (const auto& x: shapes[i])
        {
            prod *= x;
        }
        
        if (((prod + 3) >> 2) != cnt)
        {
            *__error = true;
            return {};
        }

        cnt <<= 2;
        tensors.push_back(TensorWrapper(shapes[i], data + offset));
    }

    return tensors;
}

operation_pack abi_decode_op(const int64_t* inp, uint8_t *__error)
{
    auto opcode = read_opcode(inp, __error);
    auto params = read_params(inp, __error);
    auto tensors = read_tensors(inp, __error);

    operation_pack pack = {opcode, params, tensors};
    return pack;
}

uint8_t* abi_encode_tensor(const TensorWrapper& tensor, int32_t* length)
{
    const std::vector<uint64_t>& shape = tensor.shape();
    const int64_t* data = tensor.data();

    int64_t prod = 1;
    for (const int& x: shape)
    {
        prod *= x;
    }

    int64_t padded_prod = ((prod + 3) >> 2) << 2;

    // (2 offsets, 2 counts and a list of number) * 32 bytes + data (8 bytes) * prod
    *length = ((4 + shape.size()) << 5) + (padded_prod << 3); // bytes
    int64_t short_length = *length >> 3;
    int64_t* out_int64 = new int64_t[short_length];

    std::memset(out_int64, 0, short_length << 3);
    std::memcpy(out_int64 + 12, data, prod << 3);

    out_int64[3] = 64;
    out_int64[7] = (padded_prod + 12) << 3;
    out_int64[11] = padded_prod >> 2;

    out_int64[(out_int64[7] >> 3) + 3] = shape.size();
    for (int i = 0, offset = (out_int64[7] >> 3) + 7; i < shape.size(); ++i, offset += 4)
    {
        out_int64[offset] = shape[i];
    }

    uint8_t* out = new uint8_t[*length];

    for (int i = 0, j = 0; i < short_length; ++i, j += 8)
    {
        out[j + 0] = out_int64[i] >> 56;
        out[j + 1] = out_int64[i] >> 48;
        out[j + 2] = out_int64[i] >> 40;
        out[j + 3] = out_int64[i] >> 32;
        out[j + 4] = out_int64[i] >> 24;
        out[j + 5] = out_int64[i] >> 16;
        out[j + 6] = out_int64[i] >> 8;
        out[j + 7] = out_int64[i];
    }

    delete[] out_int64;
    return out;
}


// extern "C"
uint8_t* cuda_execute_operation(
    uint8_t* payload_in, // bytes: opcode, params, shapes, tensors
    int32_t length_in, 
    int32_t* length_out,
    uint8_t* _error
)
{
    logd("cuda_execute_operation");

    int short_length = length_in >> 3;
    logd("cuda_execute_operation");

    int64_t* inp = new int64_t[short_length];
    logd("cuda_execute_operation");

    memset(inp, 0, short_length);

    logd("cuda_execute_operation");


    for (int i = 0, j = 0; i < short_length; ++i, j += 8)
    {
        logd("cuda_execute_operation loop");

        inp[i] = (int64_t(payload_in[j + 0]) << 56) 
            | (int64_t(payload_in[j + 1]) << 48) 
            | (int64_t(payload_in[j + 2]) << 40) 
            | (int64_t(payload_in[j + 3]) << 32) 
            | (int64_t(payload_in[j + 4]) << 24) 
            | (int64_t(payload_in[j + 5]) << 16) 
            | (int64_t(payload_in[j + 6]) << 8) 
            |  int64_t(payload_in[j + 7]);
    }
    logd("cuda_execute_operation");

    operation_pack pack = abi_decode_op(inp, _error);
    logd("cuda_execute_operation");

    auto wrap_return_fn = [&](uint8_t* out = nullptr) -> uint8_t* {
        delete[] inp;
        logd("wrap return fn");

        if (out == nullptr)
        {
            *_error = true;
        }

        return out;
    };

    if (*_error)
    {
        logd("cuda_execute_operation  error");
    
        return wrap_return_fn();
    }

    logd("cuda_execute_operation");
    
    if (pack.op == opcode::MATMUL)
    {
        logd("cuda_execute_operation matmul");
        return wrap_return_fn(matmul_call(pack, length_out, _error));
    }

    if (pack.op == opcode::CONV2D)
    {
        logd("cuda_execute_operation conv2d");
        return wrap_return_fn(conv2d_call(pack, length_out, _error));
    }

    if (pack.op == opcode::MAXPOOLING2D)
    {
        logd("cuda_execute_operation maxpooling2d");
        return wrap_return_fn(maxpooling2d_call(pack, length_out, _error));
    }

    if (pack.op == opcode::AVGPOOLING2D)
    {
        logd("cuda_execute_operation avgpooling2d");
        return wrap_return_fn(avgpooling2d_call(pack, length_out, _error));
    }

    if (pack.op == opcode::ELEMENTWISE_ADD)
    {
        logd("cuda_execute_operation elementwise_add");
        return wrap_return_fn(elementwise_add_call(pack, length_out, _error));
    }

    if (pack.op == opcode::ELEMENTWISE_MUL)
    {
        logd("cuda_execute_operation elementwise_mul");
        return wrap_return_fn(elementwise_mul_call(pack, length_out, _error));
    }

    if (pack.op == opcode::ELEMENTWISE_SUB)
    {
        logd("cuda_execute_operation elementwise_sub");
        return wrap_return_fn(elementwise_sub_call(pack, length_out, _error));
    }

    if (pack.op == opcode::ELEMENTWISE_DIV)
    {
        logd("cuda_execute_operation elementwise_div");
        return wrap_return_fn(elementwise_div_call(pack, length_out, _error));
    }

    if (pack.op == opcode::TRANSFORM_EXP)
    {
        logd("cuda_execute_operation transform_exp");
        return wrap_return_fn(transform_exp_call(pack, length_out, _error));
    }

    if (pack.op == opcode::TRANSFORM_SQRT)
    {
        logd("cuda_execute_operation transform_sqrt");
        return wrap_return_fn(transform_sqrt_call(pack, length_out, _error));
    }

    if (pack.op == opcode::BATCH_NORM)
    {
        logd("cuda_execute_operation batch_norm");
        return wrap_return_fn(batch_norm_call(pack, length_out, _error));
    }

    if (pack.op == opcode::LAYER_NORM)
    {
        logd("cuda_execute_operation layer_norm");
        return wrap_return_fn(layer_norm_call(pack, length_out, _error));
    }

    if (pack.op == opcode::ZSCORE)
    {
        logd("cuda_execute_operation zscore");
        return wrap_return_fn(zscore_call(pack, length_out, _error));
    }

    if (pack.op == opcode::MIN_MAX_SCALE)
    {
        logd("cuda_execute_operation min_max_scale");
        return wrap_return_fn(min_max_scale_call(pack, length_out, _error));
    }

    if (pack.op == opcode::CONCATENATE)
    {
        logd("cuda_execute_operation concatenate");
        return wrap_return_fn(concatenate_call(pack, length_out, _error));
    }

    if (pack.op == opcode::RELU)
    {
        logd("cuda_execute_operation relu");
        return wrap_return_fn(relu_call(pack, length_out, _error));
    }

    if (pack.op == opcode::TANH)
    {
        logd("cuda_execute_operation tanh");
        return wrap_return_fn(tanh_call(pack, length_out, _error));
    }

    if (pack.op == opcode::SIGMOID)
    {
        logd("cuda_execute_operation sigmoid");
        return wrap_return_fn(sigmoid_call(pack, length_out, _error));
    }

    if (pack.op == opcode::SOFTMAX)
    {
        logd("cuda_execute_operation softmax");
        return wrap_return_fn(softmax_call(pack, length_out, _error));
    }

    if (pack.op == opcode::LOGSOFTMAX)
    {
        logd("cuda_execute_operation logsoftmax");
        return wrap_return_fn(logsoftmax_call(pack, length_out, _error));
    }

    if (pack.op == opcode::SOFTMAX2D)
    {
        logd("cuda_execute_operation softmax2d");
        return wrap_return_fn(softmax2d_call(pack, length_out, _error));
    }

    if (pack.op == opcode::REDUCTION_MAX)
    {
        logd("cuda_execute_operation reduction_max");
        return wrap_return_fn(reduction_max_call(pack, length_out, _error));
    }

    if (pack.op == opcode::REDUCTION_MIN)
    {
        logd("cuda_execute_operation reduction_min");
        return wrap_return_fn(reduction_min_call(pack, length_out, _error));
    }

    if (pack.op == opcode::REDUCTION_MEAN)
    {
        logd("cuda_execute_operation reduction_mean");
        return wrap_return_fn(reduction_mean_call(pack, length_out, _error));
    }

    if (pack.op == opcode::REDUCTION_SUM)
    {
        logd("cuda_execute_operation reduction_sum");
        return wrap_return_fn(reduction_sum_call(pack, length_out, _error));
    }

    if (pack.op == opcode::REDUCTION_ARGMAX)
    {
        logd("cuda_execute_operation reduction_argmax");
        return wrap_return_fn(reduction_argmax_call(pack, length_out, _error));
    }

    if (pack.op == opcode::REDUCTION_ARGMIN)
    {
        logd("cuda_execute_operation reduction_argmin");
        return wrap_return_fn(reduction_argmin_call(pack, length_out, _error));
    }

    if (pack.op == opcode::DROPOUT)
    {
        logd("cuda_execute_operation dropout");
        return wrap_return_fn(dropout_call(pack, length_out, _error));
    }

    if (pack.op == opcode::GLOBAL_AVGPOOLING2D)
    {
        logd("cuda_execute_operation globalavgpooling");
        return wrap_return_fn(globalavgpooling_call(pack, length_out, _error));
    }

    return wrap_return_fn();
}

// extern "C"
uint8_t* cuda_execute_operation_test(
    uint8_t* payload_in, // bytes: opcode, params, shapes, tensors
    int32_t length_in, 
    int32_t* length_out,
    uint8_t* _error
)
{
    logd("cuda_execute_operation");
    *length_out = length_in;
    return payload_in;
}

// extern "C"
void deallocate_cpp_response(uint8_t* payload)
{
    if (payload != nullptr)
    {
        delete[] payload;
    }
}

// int main(int argc, char** argv)
// {
//     std::string file = argv[1];
    
//     std::ifstream inf(file, std::ios::binary);
//     inf.seekg(0, std::ios::end);
//     int nbytes = inf.tellg();
//     inf.seekg(0, std::ios::beg);

//     uint8_t* payload = new uint8_t[nbytes];

//     inf.read((char *) payload, nbytes);
    
//     bool _error = false;
//     int32_t length = 0;
//     uint8_t* output = cuda_execute_operation(payload, nbytes, &length, &_error);

//     delete[] payload;
//     delete[] output;
//     return 0;
// }
